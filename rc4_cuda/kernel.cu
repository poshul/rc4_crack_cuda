#include "hip/hip_runtime.h"
#include "rc4.h"
#include "hip/hip_runtime_api.h"

/************************************************************************/
/* 
The original idea is to obtain one key at a time, decrypt the corresponding ciphertext, and see if the resulting plaintext satisfies a certain condition.
But the process requires too many intermediate variables, and on second thought, the plaintext and ciphertext are heterogeneous or related, so the known plaintext
If the text and the ciphertext are dissimilar, we can get the value of some position of the key stream. This saves a lot of space~~
*/
/************************************************************************/

__device__ unsigned char* genKey(unsigned char* res, unsigned long long val, int* key_len)
{
	char p = maxKeyLen - 1;
	while (val&&p >=0) {
		res[p--] = (val - 1) % keyNum + start;
		val = (val - 1) / keyNum;
	}
	*key_len = (maxKeyLen - p - 1);
	return res + p + 1;
}

__global__ void crackRc4Kernel(unsigned char* key, volatile bool* found)
{
	int keyLen = 0;
	const unsigned long long totalThreadNum = gridDim.x * blockDim.x;
	const unsigned long long keyNum_per_thread = maxNum / totalThreadNum;
	unsigned long long val = (threadIdx.x + blockIdx.x * blockDim.x);
	bool justIt;
	for (unsigned long long i=0; i <= keyNum_per_thread; val += totalThreadNum, i++)
	{
		//vKey is a pointer to share_memory
		unsigned char* vKey = genKey((shared_mem + memory_per_thread * threadIdx.x), val, &keyLen);
		justIt=device_isKeyRight(vKey,keyLen,found);

		//Exit if one of the other blocks found it
		if(*found) asm("exit;");

		// the current key is not the requested one
		if (justIt)
    {
      // Find the matching key, write it to Host, save the data, modify found, and exit the program
      *found = true;
      memcpy(key, vKey, keyLen);
      key[keyLen]=0;
      __threadfence();
      asm("exit;");
      break;
    }
	}
}

void cleanup(unsigned char *key_dev, bool* found_dev)
{
  hipFree(key_dev);
  hipFree(found_dev);
  return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t crackRc4WithCuda(unsigned char* knownKeyStream_host, int knownStreamLen_host, unsigned char*key, bool*found)
{
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    return cudaStatus;
	}

	unsigned char *key_dev ;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	cudaStatus = hipMalloc((void**)&key_dev, (MAX_KEY_LENGTH + 1) * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(key_dev);
    return cudaStatus;
  }

  bool* found_dev;

	cudaStatus = hipMalloc((void**)&found_dev, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	//Check if the key variable is found
	cudaStatus = hipMemcpy(found_dev, found, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	//Copy constant memory
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(knowStream_device), knownKeyStream_host, sizeof(unsigned char) *knownStreamLen_host);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol stream failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	cudaStatus = hipMemcpyToSymbol((const void *) &knownStreamLen_device, (const void *) &knownStreamLen_host, sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol streamlen failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Launch a kernel on the GPU with one thread for each element.
	int threadNum = floor( (double) (prop.sharedMemPerBlock / MEMORY_PER_THREAD) ), share_memory = prop.sharedMemPerBlock;
	if(threadNum > MAX_THREAD_NUM )
  {
		threadNum = MAX_THREAD_NUM;
		share_memory = threadNum * MEMORY_PER_THREAD;
	}

	crackRc4Kernel<<<BLOCK_NUM, threadNum, share_memory>>>(key_dev, found_dev);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(key, key_dev, (MAX_KEY_LENGTH+1) * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(found, found_dev,  sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	return cudaStatus;
}

int main(int argc, char *argv[])
{

	unsigned char* s_box = (unsigned char*)malloc(sizeof(unsigned char)*256);

	//Key
	//unsigned char encryptKey[]="Key";

	//Load from file
  std::ifstream input_stream("cipher");
  char temp_buffer[700];
  unsigned char buffer[700];
  input_stream.read(temp_buffer,700);
  input_stream.close();
  std::strcpy(reinterpret_cast<char*>(buffer),temp_buffer);
  
  //unsigned char buffer[] = "Plaintext";
	
  int buffer_len=strlen((char*)buffer);
	
  //prepare_key(encryptKey, strlen((char*)encryptKey), s_box);
	//rc4(buffer,buffer_len,s_box);	
  
	unsigned char knownPlainText[] = "RSA2";
	int known_p_len = strlen( (char*)knownPlainText);
	unsigned char* knownKeyStream = (unsigned char*) malloc(sizeof(unsigned char) * known_p_len);
	for (int i = 0; i < known_p_len; i++)
	{
		knownKeyStream[i] = knownPlainText[i] ^ buffer[i];
	}

	unsigned char * key=(unsigned char*)malloc( sizeof(unsigned char) * (MAX_KEY_LENGTH + 1));

	hipEvent_t start,stop;
	hipError_t cudaStatus = hipEventCreate( &start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(start) failed!");
		return 1;
	}
	cudaStatus=hipEventCreate( &stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(start, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!");
		return 1;
	}

	bool found=false;
	cudaStatus = crackRc4WithCuda(knownKeyStream, known_p_len , key, &found);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(stop,0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventSynchronize failed!");
		return 1;
	}
	float useTime;
	cudaStatus = hipEventElapsedTime(&useTime,start,stop);
	useTime /= 1000;
	printf("The time we used was:%fs\n",useTime);
	if (found)
	{
		printf("The right key has been found.The right key is:%s\n",key);
    printf("%02x%02x%02x%02x%02x\n",key[0],key[1],key[2],key[3],key[4]);
		prepare_key(key, strlen( (char*)key ), s_box);
		rc4(buffer, buffer_len, s_box);
    std::ofstream outf("decrypted");
    outf.write( (char*)buffer, 700);
    outf.close();
    std::ofstream outk("outkey");
    outk.write((char*) key, 5);
    outk.close();
		printf ("\nThe clear text is:\n%s\n", buffer);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(key);
	free(knownKeyStream);
	free(s_box);
	hipDeviceReset();
	return 0;
}



