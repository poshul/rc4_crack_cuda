#include "hip/hip_runtime.h"
#include "rc4.h"

/************************************************************************/
/* 
The original idea is to obtain one key at a time, decrypt the corresponding ciphertext, and see if the resulting plaintext satisfies a certain condition.
But the process requires too many intermediate variables, and on second thought, the plaintext and ciphertext are heterogeneous or related, so the known plaintext
If the text and the ciphertext are dissimilar, we can get the value of some position of the key stream. This saves a lot of space~~
*/
/************************************************************************/

__device__ void genKey(unsigned char* keyStartAddr, unsigned long long myKeyNum, int* key_len)
{
/*	char p = maxKeyLen - 1;
	while (myKeyNum && p >=0) {
		keyStartAddr[p--] = (myKeyNum - 1) % keyNum + start;
		myKeyNum = (myKeyNum - 1) / keyNum;
	}
	*key_len = (maxKeyLen - p - 1);
	return keyStartAddr + p + 1;
*/
  size_t i = 0;
  while (myKeyNum && i < maxKeyLen)
  {
    keyStartAddr[i++] = (myKeyNum - 1) % keyNum + start;
    myKeyNum = (myKeyNum - 1) / keyNum;
  }
  *key_len = (i);
}

__global__ void crackRc4Kernel(unsigned char* key, volatile bool* found)
{
	int keyLen = 0;
	const unsigned long long totalThreadNum = gridDim.x * blockDim.x;
	const unsigned long long keyNum_per_thread = maxNum / totalThreadNum;
	unsigned long long myKeyNum = (threadIdx.x + blockIdx.x * blockDim.x);
	bool justIt;
	for (unsigned long long i=0; i <= keyNum_per_thread; myKeyNum += totalThreadNum, ++i)
	{
		//vKey is a pointer to share_memory
		unsigned char* vKey = (shared_mem + memory_per_thread * threadIdx.x);
    genKey(vKey, myKeyNum, &keyLen);
    // Pad with nulls
    while (keyLen < maxKeyLen)
    {
      vKey[keyLen++] = '\x00';
    }

		justIt=device_isKeyRight(vKey,keyLen,found);

		//Exit if one of the other blocks found it
		if(*found) asm("exit;");

		// the current key is not the requested one
		if (justIt)
    {
      // Find the matching key, write it to Host, save the data, modify found, and exit the program
      *found = true;
      memcpy(key, vKey, keyLen);
      key[keyLen]=0;
      __threadfence();
      asm("exit;");
      break;
    }
	}
}

void cleanup(unsigned char *key_dev, bool* found_dev)
{
  hipFree(key_dev);
  hipFree(found_dev);
  return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t crackRc4WithCuda(unsigned char* knownKeyStream_host, int knownStreamLen_host, unsigned char*key, bool*found)
{
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    return cudaStatus;
	}

	unsigned char *key_dev ;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	cudaStatus = hipMalloc((void**)&key_dev, (MAX_KEY_LENGTH + 1) * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(key_dev);
    return cudaStatus;
  }

  bool* found_dev;

	cudaStatus = hipMalloc((void**)&found_dev, sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	//Check if the key variable is found
	cudaStatus = hipMemcpy(found_dev, found, sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	//Copy constant memory
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(knowStream_device), knownKeyStream_host, sizeof(unsigned char) *knownStreamLen_host);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol stream failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	cudaStatus = hipMemcpyToSymbol((const void *) &knownStreamLen_device, (const void *) &knownStreamLen_host, sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol streamlen failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Launch a kernel on the GPU with one thread for each element.
	int threadNum = floor( (double) (prop.sharedMemPerBlock / MEMORY_PER_THREAD) ), share_memory = prop.sharedMemPerBlock;
	if(threadNum > MAX_THREAD_NUM )
  {
		threadNum = MAX_THREAD_NUM;
		share_memory = threadNum * MEMORY_PER_THREAD;
	}

	crackRc4Kernel<<<BLOCK_NUM, threadNum, share_memory>>>(key_dev, found_dev);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(key, key_dev, (MAX_KEY_LENGTH+1) * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(found, found_dev,  sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		cleanup(key_dev, found_dev);
    return cudaStatus;
  }

	return cudaStatus;
}

int main(int argc, char *argv[])
{

	unsigned char* s_box = (unsigned char*)malloc(sizeof(unsigned char)*256);

	//Key
	unsigned char encryptKey[host_max_key] = "KeyK";
  unsigned char buffer[] = "In cryptography, RC4 (Rivest Cipher 4, also known as ARC4 or ARCFOUR, meaning Alleged RC4, see below) is a stream cipher.";
  size_t buffer_len = strlen( (char*)buffer);
  size_t key_len = strlen( (char*)encryptKey);
  // Pad the key
  while (key_len < host_max_key)
  {
    encryptKey[key_len++] = '\x00';
  }
	
  prepare_key(encryptKey, key_len, s_box);
	rc4(buffer, buffer_len, s_box);	
  
	unsigned char knownPlainText[] = "In cr";
	int known_p_len = strlen( (char*)knownPlainText);
	unsigned char* knownKeyStream = (unsigned char*) malloc(sizeof(unsigned char) * known_p_len);
	for (int i = 0; i < known_p_len; i++)
	{
		knownKeyStream[i] = knownPlainText[i] ^ buffer[i];
	}

	unsigned char* key = (unsigned char*) malloc( sizeof(unsigned char) * (MAX_KEY_LENGTH + 1));

	hipEvent_t start,stop;
	hipError_t cudaStatus = hipEventCreate( &start);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(start) failed!");
		return 1;
	}
	cudaStatus=hipEventCreate( &stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventCreate(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(start, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(start) failed!");
		return 1;
	}

	bool found=false;
	cudaStatus = crackRc4WithCuda(knownKeyStream, known_p_len , key, &found);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cudaStatus=hipEventRecord(stop,0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventRecord(stop) failed!");
		return 1;
	}

	cudaStatus=hipEventSynchronize(stop);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipEventSynchronize failed!");
		return 1;
	}
	float useTime;
	cudaStatus = hipEventElapsedTime(&useTime,start,stop);
	useTime /= 1000;
	printf("The time we used was:%fs\n",useTime);
	if (found)
	{
		printf("The right key has been found.The right key is:%s\n",key);
		prepare_key(key, key_len, s_box);
		rc4(buffer, buffer_len, s_box);
		printf ("\nThe clear text is:\n%s\n", buffer);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	free(key);
	free(knownKeyStream);
	free(s_box);
	hipDeviceReset();
	return 0;
}



